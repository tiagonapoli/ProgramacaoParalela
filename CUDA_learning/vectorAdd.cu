#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void vadd(float* a, float *b, float *c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pulo = gridDim.x * blockDim.x;
	//printf("%d %d\n", id, pulo);
	for(int i=id;i<n;i+=pulo) {
		//printf("%d\n", i);
		c[i] = a[i] + b[i];
	}
}

inline void gera(float *x, int n) {
	for(int i=0;i<n;i++) {
		x[i] = rand()/(float)(1 + rand());
	}
}

inline void check_result(float* a, float *b, float *c, int n) {
	printf("Checking result!...\n");
	for(int i=0;i<n;i++) {
		if(fabs(a[i] + b[i] - c[i]) > 1e-5) {
			fprintf(stderr, "Erro nos calculos...");
			exit(0);
		}
	}
	printf("OK!!\n");
}

int main(int argc, char** argv) {
	
	int devID;
	hipDeviceProp_t props;

	devID = findCudaDevice(argc,(const char**) argv);
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	printf("Device %d. Name %s\n", devID, props.name);

	int n = 1 << 20;
	printf("Size %d\n", n);
	float *h_x,*h_y,*h_z;
	h_x = (float*) malloc(n*sizeof(float));
	h_y = (float*) malloc(n*sizeof(float));
	h_z = (float*) malloc(n*sizeof(float));
	if(h_z == NULL or h_x == NULL or h_y == NULL) {
		fprintf(stderr, "Deu ruim no malloc");
		exit(0);
	}

	gera(h_x,n);
	gera(h_y,n);

	float *d_x, *d_y, *d_z;
	d_x = d_y = d_z = NULL;
	checkCudaErrors(hipMalloc(&d_x, n*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_y, n*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_z, n*sizeof(float)));

	checkCudaErrors(hipMemcpy(d_x,h_x,n*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y,h_y,n*sizeof(float),hipMemcpyHostToDevice));

	int threads = 64;
	int blocks = 128;
	vadd<<<blocks,threads>>>(d_x,d_y,d_z,n);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(h_z,d_z,n*sizeof(float),hipMemcpyDeviceToHost));
	check_result(h_x,h_y,h_z,n);
	
	checkCudaErrors(hipFree(d_x));
	checkCudaErrors(hipFree(d_y));
	checkCudaErrors(hipFree(d_z));
	free(h_x);
	free(h_y);
	free(h_z);

}
