#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void reduce(float *g_idata, float *g_odata) {
	//vetor com sizeof(float) * BLOCK_SIZE
	extern __shared__ float sdata[];
	
	int tid = threadIdx.x;
	int i = blockIdx.x * 2*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	__syncthreads();

	//do reduction in shared memory
	for(int s=blockDim.x/2;s > 0; s /= 2) {
		if(tid < s) {
			sdata[tid] += sdata[tid+s]; 
		}
		__syncthreads();
	}

	if(tid == 0) g_odata[blockIdx.x] = sdata[0];
	

}

void gera(float *x, int n) {
	for(int i=0;i<n;i++) {
		x[i] = (float)i;
		//x[i] = rand()/(float)RAND_MAX;
	}
}

void check_block_sums(float *x, float *bsum, int blocks, int n) {
	printf("Blocks %d  n: %d   Total: %d\n", blocks, n, blocks*32*2);
	for(int i=0;i<blocks;i++) {
		float r = 0;
		for(int j=0;j<32*2;j++) {
			r += x[j + i*32*2];
		}
		printf("Gab: %f\nSum: %f\n\n", r, bsum[i]);
		if(fabs(bsum[i] - r) > 1e-5) {
			fprintf(stderr, "Erro no resultado!!");
			exit(0);
		}
	}

}

int main(int argc, char **argv) {
	
	int devID;
	hipDeviceProp_t props;
	devID = findCudaDevice(argc, (const char**) argv);
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	printf("Device %d - %s\n", devID, props.name);

	int n = 1 << 6;
	int threads = 32;
	int blocks = (n + 2 * threads - 1)/ (2 * threads);
	
	float *h_x, *h_block_sums, *d_x, *d_block_sums;
	h_x = (float*) calloc(n, sizeof(float));
	h_block_sums = (float*) calloc(blocks, sizeof(float));
	if(!h_x or !h_block_sums) {
		fprintf(stderr, "Deu ruim no calloc");
		exit(0);
	}

	checkCudaErrors(hipMalloc(&d_x,n*sizeof(float)));
	checkCudaErrors(hipMalloc(&d_block_sums,blocks*sizeof(float)));

	gera(h_x,n);
	checkCudaErrors(hipMemcpy(d_x,h_x,n*sizeof(float),hipMemcpyHostToDevice));

	reduce<<<blocks,threads,threads * sizeof(float)>>>(d_x,d_block_sums);
	checkCudaErrors(hipGetLastError());
	
	checkCudaErrors(hipMemcpy(h_block_sums,d_block_sums,blocks*sizeof(float),hipMemcpyDeviceToHost));
	float res = 0;
	for(int i=0;i<blocks;i++) {
		res += h_block_sums[i];
	}

	check_block_sums(h_x,h_block_sums,blocks,n);

	checkCudaErrors(hipFree(d_block_sums));
	checkCudaErrors(hipFree(d_x));
	free(h_x);
	free(h_block_sums);

}
