#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void printa() {
	printf("%d, %d\n", blockIdx.y*gridDim.x + blockIdx.x, \
					   threadIdx.z*blockDim.x*blockDim.y + threadIdx.y*blockDim.x + threadIdx.x);
}

int main(int argc, char** argv) {
	int devID;
	hipDeviceProp_t props;

	devID = findCudaDevice(argc, (const char**) argv);

	checkCudaErrors(hipGetDevice(&devID));
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	printf("%d -> %s\n", devID, props.name);

	dim3 dimGrid(2,2);
	dim3 dimBlock(2,2,2);
	printa<<<dimGrid, dimBlock>>>();
	hipDeviceSynchronize();

}
