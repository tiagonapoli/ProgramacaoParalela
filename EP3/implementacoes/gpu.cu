#include "hip/hip_runtime.h"
#include "gpu.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
typedef float __precision;

__constant__ __precision d_PI = 3.14159265359;
__device__ inline __precision f_device(int m, int k, __precision x) {
    return sin((2 * m + 1) * d_PI * x) * cos(2 * d_PI * k * x) / sin(d_PI * x);
}

__global__ void partial_sum(ll n, int m, int k, __precision *sum_block, __precision *sum2_block) {
    
	__shared__ __precision sum[GPU_THREADS];
    __shared__ __precision sum2[GPU_THREADS];

	int tid = threadIdx.x;
	//printf("%d -> %d responsible: %lld\n", threadIdx.x + blockIdx.x * blockDim.x, blockIdx.x, (n+GPU_THREADS-1LL)/GPU_THREADS);
    hiprandState state;
    hiprand_init((unsigned long long) clock() + 279LL*(threadIdx.x + blockIdx.x * blockDim.x) , 0, 0, &state);	

    __precision sum_thread = 0.;
    __precision sum2_thread = 0.;
    __precision x, f_x;
    for (ll i = 0; i < (n + GPU_THREADS - 1LL) / GPU_THREADS; i++) {
        x = hiprand_uniform(&state) / 2.0;
        f_x = f_device(m, k, x);
        sum_thread += f_x;
        sum2_thread += f_x * f_x;
    }    

    //compute block answer putting answer of each thread of the block in a 
    //array and then applying reduce at it
    sum[tid] = sum_thread;
    sum2[tid] = sum2_thread;
    __syncthreads();

    for (int i = GPU_THREADS / 2; i > 0; i = i / 2) {
        if (threadIdx.x >= i) return;
        sum[threadIdx.x] += sum[i + threadIdx.x];
        sum2[threadIdx.x] += sum2[i + threadIdx.x];
        __syncthreads();
    }

	if(tid == 0) {
        sum_block[blockIdx.x] = sum[0];
        sum2_block[blockIdx.x] = sum2[0];
	}
}

pff gpu(ll n, int m, int k, float* sum, float* sum2, ll *new_n) {
	int num_blocks = 1;
	for(ll i=2;i<=1024;i++) {
		if(n / (i * GPU_THREADS) >= 100) num_blocks = i;
	}

	ll total_threads = GPU_THREADS * num_blocks;
	*new_n = ((n + total_threads -1)/total_threads) * total_threads;
	n = *new_n;	

    __precision *d_sum_block;
    __precision *d_sum2_block;
    __precision *h_sum_block;
    __precision *h_sum2_block;
	checkCudaErrors(hipMalloc(&d_sum_block, num_blocks * sizeof(__precision)));
	checkCudaErrors(hipMalloc(&d_sum2_block, num_blocks * sizeof(__precision)));
    h_sum_block = (__precision *)calloc(num_blocks, sizeof(__precision));
    h_sum2_block = (__precision *)calloc(num_blocks, sizeof(__precision));

	checkCudaErrors(hipMemcpy(d_sum_block, h_sum_block, num_blocks * sizeof(__precision),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sum2_block, h_sum2_block, num_blocks * sizeof(__precision),hipMemcpyHostToDevice));
	partial_sum<<<num_blocks, GPU_THREADS>>>(n / num_blocks, m, k, d_sum_block, d_sum2_block);
	hipDeviceSynchronize();
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(h_sum_block, d_sum_block, num_blocks*sizeof(__precision), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_sum2_block, d_sum2_block, num_blocks*sizeof(__precision), hipMemcpyDeviceToHost));

	double aux1, aux2;
	aux1 = aux2 = 0;
    for (int i = 0; i < num_blocks; i++) {
		aux1 += h_sum_block[i];
		aux2 += h_sum2_block[i];
    }
	(*sum) = (float)aux1;
	(*sum2) = (float)aux2;


	checkCudaErrors(hipFree(d_sum_block));
	checkCudaErrors(hipFree(d_sum2_block));
	free(h_sum_block);
    free(h_sum2_block);

    return calc_res(n, *sum, *sum2);
}

void gpu_tester() {
	int tests = 10;
	ll n = 8192000000;
	float eps = 5e-3;
	printf("Testes: %d\nN: %lld\neps: %f\n\n", tests, n, eps);
	double porcentagem_acerto = testa_corretude(tests, n, eps, gpu, true);
	printf("Acerto: %lf%%\n", porcentagem_acerto * 100.0);
}

