#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

const int NUM_THREADS = 32;

__device__ inline float f_device(int m, int k, float x) {
    return sin((2 * m + 1) * PI * x) * cos(2 * PI * k * x) / sin(PI * x);
}

__global__ void partial_sum(int n, int m, int k, float *sum_block, float *sum2_block) {
	//vetor com sizeof(int) * BLOCK_SIZE
    // mutex
    //__shared__ int *mutex_block = 0;
    //__shared__ float res_block = 0;
    // reduce
    __shared__ float sum[NUM_THREADS];
    __shared__ float sum2[NUM_THREADS];

	int tid = threadIdx.x;
    hiprandState state;
    /* Each thread gets same seed, a different sequence number, no offset. */
    hiprand_init(0, NUM_THREADS * blockIdx.x + threadIdx.x, 0, &state);	

    float sum_thread = 0.;
    float sum2_thread = 0.;
    float x, f_x;
    for (int i = 0; i < (n + NUM_THREADS - 1) / NUM_THREADS; i++) {
        x = hiprand_uniform() / 2.0;
        f_x = f_device(m, k, x);
        sum_thread += f_x;
        sum2_thread += f_x * f_x;
    }    

    // compute block answer using mutex
    /*while(atomicCAS(mutex_block,0,1) != 0);  //lock
    res_block += calc_res_device(n, sum_thread, sum2_thread);
    atomicExch(mutex_block, 0);  //unlock*/
	
    //compute block answer putting answer of each thread of the block in a 
    //array and then applying reduce at it
    sum[tid] = sum_thread;
    sum2[tid] = sum2_thread;
    __syncthreads();

    for (int i = NUM_THREADS / 2; i > 0; i = i / 2) {
        if (threadIdx.x >= i) return;
        sum[threadIdx.x] += sum[i + threadIdx.x];
        sum2[threadIdx.x] += sum2[i + threadIdx.x];
        __syncthreads();
    }

	if(tid == 0) {
        // atomic
		//block_res[blockIdx.x] = res_block;
        // reduce 
        sum_block[blockIdx.x] = sum[0];
        sum2_block[blockIdx.x] = sum2[0];
	}
}

pff gpu(ll n, int m, int k) {
	int num_blocks = (n + 2 * threads - 1)/ (2 * threads);
	
    float *d_sum_block;
    float *d_sum2_block;
    float *h_sum_block;
    float *h_sum2_block;
	checkCudaErrors(hipMalloc(&d_sum_block, num_blocks * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_sum2_block, num_blocks * sizeof(float)));
    h_sum_block = (float *)calloc(num_blocks, sizeof(float));
    h_sum2_block = (float *)calloc(num_blocks, sizeof(float));

	checkCudaErrors(hipMemcpy(d_sum_block, h_sum_block, num_blocks * sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sum2_block, h_sum2_block, num_blocks * sizeof(float),hipMemcpyHostToDevice));
	partial_sum<<<num_blocks, NUM_THREADS>>>((n + num_blocks - 1) / num_blocks, m, k, d_sum_block, d_sum2_block);
	hipDeviceSynchronize();
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(h_sum_block, d_sum_block, num_blocks*sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_sum2_block, d_sum2_block, num_blocks*sizeof(float), hipMemcpyDeviceToHost));

    float sum = 0.;
    float sum2 = 0.;
    for (int i = 0; i < num_blocks; i++) {
        sum += h_sum_block[i];
        sum2 += h_sum2_block[i];
    }

	checkCudaErrors(hipFree(d_sum_block));
	checkCudaErrors(hipFree(d_sum2_block));
	free(h_sum_block);
    free(h_sum2_block);

    return calc_res(n, sum, sum2);
}

int main() {
    double porcentagem_acerto = testa_corretude(50, 32000000, 5e-3, gpu, true);
    printf("Acerto: %lf%%\n", porcentagem_acerto * 100.0);
}
