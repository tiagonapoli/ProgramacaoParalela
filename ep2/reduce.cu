#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "utils.cpp"

__device__ inline int min(int a, int b) {
	if(a > b) {
		return a;
	}
	return b;
}

__global__ void reduce(int *in, int *out) {
	//vetor com sizeof(int) * BLOCK_SIZE
	extern __shared__ int sdata[];
	
	int tid = threadIdx.x;
	int i = blockIdx.x * 2*blockDim.x + threadIdx.x;
	sdata[tid] = in[i] + in[i + blockDim.x];
	__syncthreads();

	//do reduction in shared memory
	for(int s=blockDim.x/2;s > 0; s /= 2) {
		if(tid < s) {
			sdata[tid] = min(sdata[tid], sdata[tid+s]); 
		}
		__syncthreads();
	}

	if(tid == 0) {
		out[blockIdx.x] = sdata[0];
	}
}

int solve_for_position(int n, int blocks, int *h_x, int *h_block_min, int *d_x, int *d_block_min) {
	checkCudaErrors(hipMemcpy(d_x,h_x,n*sizeof(int),hipMemcpyHostToDevice));
	reduce<<<blocks,threads,threads * sizeof(int)>>>(d_x,d_block_min);
	checkCudaErrors(hipGetLastError());
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(h_block_min,d_block_min,blocks*sizeof(int),hipMemcpyDeviceToHost));
	int res = h_block_min[0];
	for(int i=0;i<blocks;i++) {
		res = min(res, h_block_min[i]);
	}
	return res;
}

int main(int argc, char **argv) {
	
	int devID;
	hipDeviceProp_t props;
	devID = findCudaDevice(argc, (const char**) argv);
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	int n;
	int ***h_x = read(argc, argv, &n);
	
	int threads = 32;
	int blocks = (n + 2 * threads - 1)/ (2 * threads);
	
	int *h_block_min, *d_x, *d_block_min;
	h_block_min = (int*) calloc(blocks, sizeof(int));
	if(!h_x or !h_block_min) {
		fprintf(stderr, "Deu ruim no calloc");
		exit(0);
	}

	checkCudaErrors(hipMalloc(&d_x,n*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_block_min,blocks*sizeof(int)));

	int res[3][3];

	for(int i=0;i<3;i++) {
		for(int j=0;j<3;j++) {
			res[i][j] = solve_for_position(n,blocks,h_x[i][j],h_block_min,d_x,d_block_min);
		}
	}

	print_matrix(res);

	checkCudaErrors(hipFree(d_block_min));
	checkCudaErrors(hipFree(d_x));
	free(h_x);
	free(h_block_min);

}
