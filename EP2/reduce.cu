#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "utils.cpp"
#include <iostream>
using namespace std;
bool debug = 0;
#define prin(a) if(debug) cout << #a << ": " << (a) << endl


int threads = 32;

__device__ inline int min_cuda(int a, int b) {
	if(a < b) {
		return a;
	}
	return b;
}

__global__ void reduce(int *in, int *out) {
	//vetor com sizeof(int) * BLOCK_SIZE
	extern __shared__ int sdata[];
	
	int tid = threadIdx.x;
	int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	sdata[tid] = min_cuda(in[i], in[i + blockDim.x]);
	__syncthreads();

	//do reduction in shared memory
	for(int s=blockDim.x/2;s > 0; s /= 2) {
		if(tid < s) {
			sdata[tid] = min_cuda(sdata[tid], sdata[tid+s]); 
		}
		__syncthreads();
	}

	if(tid == 0) {
		out[blockIdx.x] = sdata[0];
	}
}

int solve_for_position(int n, int blocks, int *h_x, int *h_block_min, int *d_x, int *d_block_min) {

	checkCudaErrors(hipMemcpy(d_x,h_x,n*sizeof(int),hipMemcpyHostToDevice));
	reduce<<<blocks,threads,threads * sizeof(int)>>>(d_x,d_block_min);
	hipDeviceSynchronize();
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemcpy(h_block_min,d_block_min,blocks*sizeof(int),hipMemcpyDeviceToHost));
	
	int res = h_block_min[0];
	for(int i=0;i<blocks;i++) {
		res = min_seq(res, h_block_min[i]);
	}
	return res;

}

int main(int argc, char **argv) {
	
	int devID;
	hipDeviceProp_t props;
	devID = 0;
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	int n;
	int ***h_x = read(argc, argv, &n);
	
	int blocks = (n + 2 * threads - 1)/ (2 * threads);
	
	int *h_block_min, *d_x, *d_block_min;
	h_block_min = (int*) calloc(blocks, sizeof(int));
	if(!h_x or !h_block_min) {
		fprintf(stderr, "Deu ruim no calloc");
		exit(0);
	}

	checkCudaErrors(hipMalloc(&d_x,n*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_block_min,blocks*sizeof(int)));

	int res[3][3];

	clock_t t1 = clock();
	for(int i=0;i<3;i++) {
		for(int j=0;j<3;j++) {
			res[i][j] = solve_for_position(n,blocks,h_x[i][j],h_block_min,d_x,d_block_min);
		}
	}
	clock_t t2 = clock();
	
	FILE* f = fopen("time_cuda.out", "w");
	fprintf(f, "%.0lf ", (double)(t2-t1) * 1000000/CLOCKS_PER_SEC);
	fclose(f);

	print_matrix(res);

	checkCudaErrors(hipFree(d_block_min));
	checkCudaErrors(hipFree(d_x));
	free(h_x);
	free(h_block_min);

}
