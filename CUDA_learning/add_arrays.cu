
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void add(int n, float* x, float* y) {
    for(int i=0;i<n;i++) {
        y[i] = x[i] + y[i];
    }
}

int main() {
    int n = 1 << 20;

    float *x = new float[n];
    float *y = new float[n];

    for(int i=0;i<n;i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add(n,x,y);

    float error = 0.0f;
    for(int i=0;i<n;i++) {
        error = fmax(error, fabs(y[i]-3.0f));
    }
    printf("Max error: %f\n", error);

    delete [] x;
    delete [] y;   

    return 0;
}