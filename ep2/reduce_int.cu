#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__device__ inline int min(int a, int b) {
	if(a > b) {
		return a;
	}
	return b;
}

__global__ void reduce(int *in, int *out) {
	//vetor com sizeof(int) * BLOCK_SIZE
	extern __shared__ int sdata[];
	
	int tid = threadIdx.x;
	int i = blockIdx.x * 2*blockDim.x + threadIdx.x;
	sdata[tid] = in[i] + in[i + blockDim.x];
	__syncthreads();

	//do reduction in shared memory
	for(int s=blockDim.x/2;s > 0; s /= 2) {
		if(tid < s) {
			sdata[tid] = min(sdata[tid], sdata[tid+s]); 
		}
		__syncthreads();
	}

	if(tid == 0) {
		out[blockIdx.x] = sdata[0];
	}
}

inline int min(int a, int b) {
	return a > b ? a : b;
}

int*** read(char *path, int &n) {
	FILE* f = fopen(path, "r");
	
	fscanf(f, "%d", &n);
	
	int*** x;
	x = (int***) malloc(3*sizeof(int**));
	for(int i=0;i<3;i++) {
		x[i] = (int**) malloc(3*sizeo(int*));
		for(int j=0;j<3;j++) {
			x[i][j] = (int*) malloc(n * sizeof(int));
		}
	}

	for(int i=0;i<n;i++) {
		fscanf(f, "%*s");
		for(int j=0;j<3;j++) {
			for(int k=0;k<3;k++) {
				fscanf(f, "%d", &x[j][k][i]);
			}
		}
	}

	fclose(f);
	return x;
}

int solve(int **x, int n) {
	int res = x[0];
	for(int i=0;i<n;i++) {
		res = min(res, x[i]);
	}
	return res;
}

int solve_for_position(int n, int blocks, int *h_x, int *h_block_min, int *d_x, int *d_block_min) {
	
	checkCudaErrors(hipMemcpy(d_x,h_x,n*sizeof(int),hipMemcpyHostToDevice));

	reduce<<<blocks,threads,threads * sizeof(int)>>>(d_x,d_block_min);
	checkCudaErrors(hipGetLastError());
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(h_block_min,d_block_min,blocks*sizeof(int),hipMemcpyDeviceToHost));
	int res = 0;
	for(int i=0;i<blocks;i++) {
		res += h_block_min[i];
	}
	return res;
}

int main(int argc, char **argv) {
	
	int devID;
	hipDeviceProp_t props;
	devID = findCudaDevice(argc, (const char**) argv);
	checkCudaErrors(hipGetDeviceProperties(&props, devID));

	int n = 1 << 22;
	int threads = 32;
	int blocks = (n + 2 * threads - 1)/ (2 * threads);
	
	int *h_x, *h_block_min, *d_x, *d_block_min;
	h_x = (int*) calloc(n, sizeof(int));
	h_block_min = (int*) calloc(blocks, sizeof(int));
	if(!h_x or !h_block_min) {
		fprintf(stderr, "Deu ruim no calloc");
		exit(0);
	}
	checkCudaErrors(hipMalloc(&d_x,n*sizeof(int)));
	checkCudaErrors(hipMalloc(&d_block_min,blocks*sizeof(int)));



	checkCudaErrors(hipFree(d_block_min));
	checkCudaErrors(hipFree(d_x));
	free(h_x);
	free(h_block_min);

}
