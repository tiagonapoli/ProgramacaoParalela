
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void add(int n, float* x, float* y) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int pulo = gridDim.x * blockDim.x;
	for(int i=tid;i < n; i += pulo) {
		y[i] = x[i] + y[i];
	}
}


int main() {
    int n = 1 << 25;
	
    float *x,*y;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));


    for(int i=0;i<n;i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

	int block_size = 128;
	int num_blocks = 4096;
    add<<<num_blocks, block_size>>>(n,x,y);

    hipDeviceSynchronize();

    float error = 0.0f;
    for(int i=0;i<n;i++) {
        error = fmax(error, fabs(y[i]-3.0f));
    }
    printf("Max error: %f\n", error);

    hipFree(x);
    hipFree(y);

    return 0;
}
